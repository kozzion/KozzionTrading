
#include <hip/hip_runtime.h>

// dimensions layout: 
// 0-2 target offset_size        (flat)
// 3-5 source and inclusion size (incremental)
// 6-8 filter size               (not incremental)
// 9   sample_count
__global__ void cuda_filter_tips(
	float *target_image,  
	float const *const source_image,
    float const *const inclusion_image, 	
    float const *const filter_kernel,  
    int const *const dimensions,	 
    float const *const error_measure,
	unsigned int const *const real_grid_dimensions)
{
    //indexing
	unsigned int block_index_x           = blockIdx.x % real_grid_dimensions[0]; 
	unsigned int block_index_y           = blockIdx.x / real_grid_dimensions[0]; 
    unsigned int block_index_z           = blockIdx.y;
	unsigned int target_index_x          = threadIdx.x + (block_index_x * blockDim.x); 
    unsigned int target_index_y          = threadIdx.y + (block_index_y * blockDim.y);
	unsigned int target_index_z          = threadIdx.z + (block_index_z * blockDim.z);

	unsigned int target_index_frame_offset =   target_index_x + dimensions[0]+
		                                     ((target_index_y + dimensions[1]) * dimensions[3])+
								             ((target_index_z + dimensions[2]) * dimensions[4]);

    float total_contribution              = 0;
    for (int filter_index_x = 0; filter_index_x < dimensions[6]; filter_index_x++)
    {
        for (int filter_index_y = 0; filter_index_y < dimensions[7]; filter_index_y++)
        {
             for (int filter_index_z = 0; filter_index_z < dimensions[8]; filter_index_z++)
             {
                // we can precompute these index offsets moving them to a single loop, (speeup^^)
                int filter_index = filter_index_x +
                                  (filter_index_y * dimensions[6]) +
                                  (filter_index_z * dimensions[6] * dimensions[7]);  

                int source_index_frame_offset = (target_index_x + filter_index_x) +
                                               ((target_index_y + filter_index_y) * dimensions[3]) +
                                	           ((target_index_z + filter_index_z) * dimensions[4]);

                // compute error, this is were the magic happens: 
                // choose any function that has range {0, 1} to determine contribution
                float mse = 0;
                for (int time_index = 0; time_index < dimensions[9]; time_index++)
                {
                    float time_error = source_image[target_index_frame_offset + (time_index * dimensions[5])] - 
                                       source_image[source_index_frame_offset + (time_index * dimensions[5])];
                    mse += time_error * time_error;
                }

                //this next bit can be done beforehand

                float k = (mse * mse)  / (43245000.0f);
                if (k >= 4.61f) 
                {
                	continue;
                }

                //float contribution = filter_kernel[filter_index] * inclusion_image[source_index_frame_offset];
                float contribution = filter_kernel[filter_index] * inclusion_image[source_index_frame_offset] * expf(-k);
                //float contribution = filter_kernel[filter_index] * expf(-k);

                // here the magic ends
                total_contribution += contribution;
                //total_contribution += mse * filter_kernel[filter_index];
                for (int time_index = 0; time_index < dimensions[9]; time_index++)
                {
                    target_image[target_index_frame_offset + (time_index * dimensions[5])] += source_image[source_index_frame_offset + (time_index * dimensions[5])] * contribution;
                }
            }
        }
    }
    for (int time_index = 0; time_index < dimensions[9]; time_index++)
    {
        target_image[target_index_frame_offset + (time_index * dimensions[5])] /= total_contribution;
        //target_image[target_index_frame_offset + (time_index * dimensions[2])] = total_contribution;
    }
}