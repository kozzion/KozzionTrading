
#include <hip/hip_runtime.h>

// dimensions layout: 
// 0-2 target offset_size        (flat)
// 3-5 source and inclusion size (incremental)
// 6-8 filter size               (not incremental)
// 9   sample_count
__global__ void cuda_filter_tips_fast(
	float *target_image,  
	float const *const source_image,
    float const *const inclusion_image, 	
    float const *const filter_kernel,      
    int const *const dimensions,	
    int const *const filter_source_offsets,  
    float const *const error_measure,
	unsigned int const *const real_grid_dimensions)
{
    //indexing
	unsigned int block_index_x           = blockIdx.x % real_grid_dimensions[0]; 
	unsigned int block_index_y           = blockIdx.x / real_grid_dimensions[0]; 
    unsigned int block_index_z           = blockIdx.y;
	unsigned int target_index_x          = threadIdx.x + (block_index_x * blockDim.x); 
    unsigned int target_index_y          = threadIdx.y + (block_index_y * blockDim.y);
	unsigned int target_index_z          = threadIdx.z + (block_index_z * blockDim.z);

	unsigned int target_index_frame_offset =   target_index_x + dimensions[0]+
		                                     ((target_index_y + dimensions[1]) * dimensions[3])+
								             ((target_index_z + dimensions[2]) * dimensions[4]);

    float total_contribution              = 0;
    for (int filter_index = 0; filter_index < dimensions[6]; filter_index++)
    {
        int source_index_frame_offset = target_index_frame_offset + filter_source_offsets[filter_index];
         if (inclusion_image[source_index_frame_offset] == 0.0f) 
        {
            continue;
        }
        // compute error, this is were the magic happens: 
        // choose any function that has range {0, 1} to determine contribution
        float mse = 0;
        for (int time_index = 0; time_index < dimensions[9]; time_index++)
        {
            float time_error = source_image[target_index_frame_offset + (time_index * dimensions[5])] - 
                               source_image[source_index_frame_offset + (time_index * dimensions[5])];
            mse += time_error * time_error;
        }
        // note magic number
        float k = (mse * mse)  / (43245000.0f);
        if (k >= 4.61f) 
        {
            continue;
        }

        float contribution = filter_kernel[filter_index] * inclusion_image[source_index_frame_offset] * expf(-k);

        // here the magic ends
        total_contribution += contribution;
        for (int time_index = 0; time_index < dimensions[9]; time_index++)
        {
            target_image[target_index_frame_offset + (time_index * dimensions[5])] += source_image[source_index_frame_offset + (time_index * dimensions[5])] * contribution;
        }
     
    }
    for (int time_index = 0; time_index < dimensions[9]; time_index++)
    {
        target_image[target_index_frame_offset + (time_index * dimensions[5])] /= total_contribution;

    }
}